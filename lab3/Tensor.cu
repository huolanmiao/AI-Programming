#include "Tensor.h"
#include <iostream>
#include <algorithm>
#include <cmath>
#include <hip/hip_runtime.h>
#include <cstddef>

Tensor::Tensor(const std::vector<int>& shape, const std::string & device):
        shape(shape), device(device) {
        int totalSize = 1;
        for (int dim : shape) totalSize *= dim;
        if (device == "GPU") {
            hipError_t err = hipMalloc(&data, totalSize*sizeof(float));  // Allocate memory on GPU
            // printf("GPU_cudaMalloc: %s\n", hipGetErrorString(err));
        } else {
            data = new float[totalSize];  // Allocate memory on CPU
        }
}

Tensor::Tensor(const Tensor& other) {
    if (data) {
        if (device == "GPU") hipFree(data);
        else delete[] data;
    }
    this->shape = other.shape;
    this->device = other.device;
    int totalSize = 1;
    for (int dim : shape) totalSize *= dim;

    if (device == "GPU") {
        hipMalloc(&data, totalSize * sizeof(float));  // Allocate memory on GPU
        hipMemcpy(data, other.data, totalSize * sizeof(float), hipMemcpyDeviceToDevice);  // Copy data to GPU
    } else {
        this->data = new float[totalSize];  // Allocate memory on CPU
        std::copy(other.data, other.data + totalSize, data);  // Copy data to CPU
    }
}

Tensor::Tensor(py::array_t<float> new_data, const std::string & device): device(device){
    shape = std::vector<int>(new_data.ndim());
    for (int i = 0; i<shape.size(); ++i){
        shape[i] = new_data.shape(i);
    }
    // Allocate memory and Store data
    int totalSize = 1;
    for (int dim : shape) totalSize *= dim;
    if (device == "GPU") {
        hipError_t err = hipMalloc(&data, totalSize*sizeof(float));  // Allocate memory on GPU
        // printf("GPU_cudaMalloc: %s\n", hipGetErrorString(err));
        hipMemcpy(data, new_data.data(), totalSize * sizeof(float), hipMemcpyHostToDevice);
    } else {
        data = new float[totalSize];  // Allocate memory on CPU
        std::copy(new_data.data(), new_data.data() + totalSize, data); 
    }
}

py::array_t<float> Tensor::to_numpy(){
    int totalSize = 1;
    for (int dim : shape) totalSize *= dim;
    auto a = py::array_t<float>(totalSize);
    py::buffer_info buf = a.request();
    float* ptr = (float*)buf.ptr; 
    if(device == "GPU"){
        hipMemcpy(ptr, data, totalSize * sizeof(float), hipMemcpyDeviceToHost);
    } else{
        std::copy(data, data + totalSize, ptr);
    }
    a.resize(shape);
    return a;
}


void Tensor::set_data(py::array_t<float> new_data){
    // Correct the shape
    int original_size = 1;
    for(int dim : shape) original_size *= dim;

    shape = std::vector<int>(new_data.ndim());
    for (int i = 0; i<shape.size(); ++i){
        shape[i] = new_data.shape(i);
    }
    // Allocate memory and Store data
    int totalSize = 1;
    for (int dim : shape) totalSize *= dim;
    if (totalSize != original_size){
        printf("Reallocate memory!");
        if (device == "GPU") {
            hipFree(data);  // Free GPU memory
            hipError_t err = hipMalloc(&data, totalSize*sizeof(float));  // Allocate memory on GPU
            printf("GPU_cudaMalloc: %s\n", hipGetErrorString(err));
        } else {
            delete[] data;  // Free CPU memory
            data = new float[totalSize];  // Allocate memory on CPU
        }
    }
    if (device == "GPU") {
        hipMemcpy(data, new_data.data(), totalSize * sizeof(float), hipMemcpyHostToDevice);
    } else {
        for (int i = 0; i<totalSize; ++i){
            data[i] = new_data.data()[i];
        }
    }
    
}

void Tensor::show_tensor() const{
    // 打印shape和device
    printf("shape: [");
    for (int i = 0; i < shape.size(); i++) {
        printf("%d", shape[i]);
        if (i < shape.size() - 1) printf(", ");
    }
    printf("]\n");
    printf("device: %s\n", device);
    int totalSize = 1;
    for (int dim : shape) totalSize *= dim;
    if (device == "GPU") {
        // 打印GPU上的数据
        float* cpu_data = new float[totalSize];
        hipMemcpy(cpu_data, data, totalSize * sizeof(float), hipMemcpyDeviceToHost);
        if (shape.size() == 2){
            for(int i = 0; i < shape[0]; i++) {
                for(int j = 0; j < shape[1]; j++) {
                    printf("%f ", cpu_data[i*shape[1] + j]);
                }
                printf("\n");
            }
        } else{
            for(int i = 0; i<totalSize; i++){
                printf("%f ", cpu_data[i]);
            }
            printf("\n");
        }
        delete[] cpu_data;
    } else {
        // 打印CPU上的数据
        if(shape.size() == 2){
            for(int i = 0; i < shape[0]; i++) {
                for(int j = 0; j < shape[1]; j++) {
                    printf("%f ", data[i*shape[1] + j]);
                }
                printf("\n");
            }
        } else{
            for(int i = 0; i<totalSize; i++){
                printf("%f ", data[i]);
            }
            printf("\n");
        }
        
    }
        
}


// 析构函数
Tensor::~Tensor() {
    if (device == "GPU") {
        hipFree(data);  // Free GPU memory
    } else {
        delete[] data;  // Free CPU memory
    }
}

// 将数据复制到 CPU
Tensor Tensor::cpu() {
    // Copy data to CPU 
    if (device == "CPU") return *this;
    else {
        // printf("Data coping to CPU.\n");
        int totalSize = 1;
        for (int dim : shape) totalSize *= dim;
        float* cpu_data = new float[totalSize];  // Allocate memory on CPU
        hipError_t err_cpy = hipMemcpy(cpu_data, this->data, totalSize*sizeof(float), hipMemcpyDeviceToHost);  // Copy data to CPU
        // printf("hipMemcpy: %s\n", hipGetErrorString(err_cpy));
        hipError_t err_free = hipFree(this->data);  // Free GPU memory
        // printf("hipFree: %s\n", hipGetErrorString(err_free));
        this->data = cpu_data;
        this->device = "CPU";
        // printf("Data copied to CPU.\n\n");
        return *this;
    }        
}
// 将数据复制到 GPU
Tensor Tensor::gpu() {
    // Copy data to GPU 
    // std::cout << "Data copied to GPU." << std::endl;
    if (device == "GPU") return *this;
    else {
        // printf("Data coping to GPU.\n");
        int totalSize = 1;
        for (int dim : shape) totalSize *= dim;
        float* gpu_data;
        hipError_t err_malloc = hipMalloc(&gpu_data, totalSize*sizeof(float));  // Allocate memory on GPU
        // printf("hipMalloc: %s\n", hipGetErrorString(err_malloc));
        hipError_t err_cpy = hipMemcpy(gpu_data, this->data, totalSize*sizeof(float), hipMemcpyHostToDevice);  // Copy data to GPU
        // printf("hipMemcpy: %s\n", hipGetErrorString(err_cpy));
        delete[] this->data;  // Free CPU memory
        this->data = gpu_data;
        this->device = "GPU";
        // printf("Data copied to GPU.\n\n");
        return *this;
    }        
}
